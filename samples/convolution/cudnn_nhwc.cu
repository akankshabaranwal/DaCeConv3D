// References:
// http://courses.cms.caltech.edu/cs101gpu/2022_lectures/cs179_2022_lec17.pdf
// https://gist.github.com/odashi/1c20ba90388cf02330e1b95963d78039
// https://medium.com/@rohitdwivedula/minimal-cudnn-c-hello-world-example-47d3c6b60b73
// API Reference: https://docs.nvidia.com/deeplearning/cudnn/api/index.html#cudnnConvolutionForward
// https://gist.github.com/goldsborough/865e6717e64fbae75cdaf6c9914a130d

#include <iomanip>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <cassert>
#include <iostream>
#include <algorithm>

using namespace std;

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << ' ' << hipGetErrorString(err) <<' '<< __LINE__ << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  cudnnStatus_t err = (f); \
  if (err != CUDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << ' '<< cudnnGetErrorString(err)<<' '<<__LINE__<< std::endl; \
    std::exit(1); \
  } \
}

int main()
{
        int numGPUs;
        hipGetDeviceCount(&numGPUs);
        std::cout << "Found " << numGPUs << " GPUs." << std::endl;
        hipSetDevice(0); // use GPU0
        int device; 
        struct hipDeviceProp_t devProp;
        hipGetDevice(&device);
        hipGetDeviceProperties(&devProp, device);
        std::cout << "Compute capability:" << devProp.major << "." << devProp.minor << std::endl;

        cudnnHandle_t cudnn;
        CUDNN_CALL(cudnnCreate(&cudnn));
        std::cout << "Created cuDNN handle" << std::endl;

        // input
        const int in_n = 4, in_c = 4, in_d = 128, in_h = 128, in_w = 128;
        std::cout << "in_n: " << in_n << ", in_c: " << in_c << ", in_d: " << in_d << ", in_h: " << in_h << ", in_w: " << in_w << std::endl;
        cudnnTensorDescriptor_t in_desc;
        CUDNN_CALL(cudnnCreateTensorDescriptor(&in_desc));
        int dims[5];
        dims[0]=in_n; dims[1]=in_c; dims[2]=in_d; dims[3]=in_h; dims[4]=in_w; 
        //dims[0]=in_n; dims[1]=in_d; dims[2]=in_h; dims[3]=in_w; dims[4]=in_c; 
        //int strides[5];
        //strides[0]=in_c*in_d*in_h*in_w; strides[1]=in_d*in_h*in_w; strides[2]=in_h*in_w; strides[3]=in_w; strides[4]=1;
        //strides[0]=in_d*in_h*in_w*in_c; strides[1]=in_h*in_w*in_c; strides[2]=in_w*in_c; strides[3]=in_c; strides[4]=1;
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(in_desc,
                                        CUDNN_TENSOR_NHWC,
                                        CUDNN_DATA_FLOAT,
                                        5,
                                        dims));
        float *in_data;
        CUDA_CALL(hipMalloc( &in_data, in_n * in_c * in_d * in_h * in_w * sizeof(float)));

        // filter
        const int filt_k = 1, filt_c = 1, filt_d = 3, filt_h = 3, filt_w = 3;
        std::cout << "filt_k: " << filt_k << ", filt_c: " << filt_c << ", filt_d: " << filt_d << ", filt_h: " << filt_h << ", filt_w: " << filt_w << std::endl;
        cudnnFilterDescriptor_t filt_desc;
        CUDNN_CALL(cudnnCreateFilterDescriptor(&filt_desc));
        vector<int> filtdims = {filt_k, in_c, filt_d, filt_h, filt_w};
        //vector<int> filtdims = {filt_k, filt_d, filt_h, filt_w, in_c};
        CUDNN_CALL(cudnnSetFilterNdDescriptor(filt_desc, 
                                              CUDNN_DATA_FLOAT, 
                                              CUDNN_TENSOR_NHWC,
                                              5, 
                                              filtdims.data()));
        float *filt_data;
        CUDA_CALL(hipMalloc(&filt_data, filt_k * filt_c * filt_d * filt_h * filt_w * sizeof(float)));

        // convolution
        const int pad_d = 0, pad_h = 0, pad_w = 0, str_d=1, str_h = 1, str_w = 1, dil_d=1, dil_h = 1, dil_w = 1;
        std::cout << "pad_d: " << pad_d << ", pad_h: " << pad_h << ", pad_w: " << pad_w << ", str_d: " << str_d << ", str_h: " << str_h << ", str_w: " << str_w << ", dil_d: " << dil_d <<", dil_h: " << dil_h << ", dil_w: " << dil_w << std::endl;
        cudnnConvolutionDescriptor_t conv_desc;
        CUDNN_CALL(cudnnCreateConvolutionDescriptor(&conv_desc));
        vector<int> convpad = {pad_d, pad_h, pad_w};
        vector<int> filtstr = {str_d, str_h, str_w};
        vector<int> convdil = {dil_d, dil_h, dil_w};
        CUDNN_CALL(cudnnSetConvolutionNdDescriptor(conv_desc, 
                                                  3, 
                                                  convpad.data(), 
                                                  filtstr.data(), 
                                                  convdil.data(), 
                                                  CUDNN_CROSS_CORRELATION, 
                                                  CUDNN_DATA_FLOAT));

        // output
        int outdims[5];
        CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim( conv_desc,
                                                        in_desc,
                                                        filt_desc,
                                                        5,
                                                        outdims));

        cudnnTensorDescriptor_t out_desc;        
        CUDNN_CALL(cudnnCreateTensorDescriptor(&out_desc));
        int out_n = outdims[0]; int out_c = outdims[1]; int out_d = outdims[2]; int out_h = outdims[3]; int out_w = outdims[4];
        //int out_n = outdims[0]; int out_d = outdims[1]; int out_h = outdims[2]; int out_w = outdims[3]; int out_c = outdims[4];

        std::cout << "out_n: " << out_n << ", out_c: " << out_c << ", out_d: "<< out_d<< ", out_h: " << out_h << ", out_w: " << out_w << std::endl;
        //vector<int> outstrides = {out_c*out_d*out_h*out_w, out_d*out_h*out_w, out_h*out_w, out_w, 1};
        vector<int> outstrides = { out_d*out_h*out_w*out_c, out_h*out_w*out_c, out_w*out_c, out_c, 1};
        
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(out_desc, 
                                              CUDNN_TENSOR_NHWC,
                                              CUDNN_DATA_FLOAT,
                                              5,
                                              outdims));
        float *out_data;
        CUDA_CALL(hipMalloc(&out_data, out_n * out_c * out_d * out_h * out_w * sizeof(float)));

        void *search_ws;
        hipMalloc(&search_ws, 33554432);        
        cudnnConvolutionFwdAlgoPerf_t perfResults[CUDNN_CONVOLUTION_FWD_ALGO_COUNT];
        int returnedAlgoCount;
        CUDNN_CALL(cudnnFindConvolutionForwardAlgorithmEx(cudnn, 
                                                        in_desc,
                                                        in_data,
                                                        filt_desc, 
                                                        filt_data, 
                                                        conv_desc, 
                                                        out_desc, 
                                                        out_data, 
                                                        CUDNN_CONVOLUTION_FWD_ALGO_COUNT, 
                                                        &returnedAlgoCount, 
                                                        perfResults, 
                                                        search_ws, 
                                                        33554432));
      for (int i=0; i<returnedAlgoCount; i++)
        std::cout<<perfResults[i].status<<' '<<perfResults[i].algo<<std::endl;


      // Till here the code works.
        assert(in_desc!=nullptr);
        assert(filt_desc!=nullptr);
        assert(out_desc!=nullptr);
        assert(conv_desc!=nullptr);

        hipFree(search_ws);
        
        cudnnConvolutionFwdAlgo_t selectedAlgo;
        //selectedAlgo = perfResults.algo;
        selectedAlgo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
        std::cout<<selectedAlgo;

        //return 0;
        
        size_t ws_size=3355443200;
        CUDNN_CALL(cudnnGetConvolutionForwardWorkspaceSize(cudnn, 
                                                          in_desc, 
                                                          filt_desc, 
                                                          conv_desc, 
                                                          out_desc, 
                                                          selectedAlgo, 
                                                          &ws_size));
        std::cerr << "Workspace size: " << (ws_size ) << "bytes"<< std::endl;
        
        void* d_workspace{nullptr};
        hipMalloc(&d_workspace, ws_size);
        const float alpha = 1.0f, beta = 0.0f;
        CUDNN_CALL(cudnnConvolutionForward(cudnn, 
                                        &alpha, 
                                        in_desc, 
                                        in_data, 
                                        filt_desc, 
                                        filt_data, 
                                        conv_desc, 
                                        selectedAlgo, 
                                        d_workspace, 
                                        ws_size, 
                                        &beta, 
                                        out_desc, 
                                        out_data));
        hipFree(in_data);
        hipFree(out_data);
        hipFree(filt_data);
        hipFree(d_workspace);
        cudnnDestroyTensorDescriptor(in_desc);
        cudnnDestroyTensorDescriptor(out_desc);
        cudnnDestroyFilterDescriptor(filt_desc);
        cudnnDestroyConvolutionDescriptor(conv_desc);
}

// Command line to compile: nvcc cudnn_conv.cu -I /users/abaranwa/cudnn-linux-x86_64-8.4.1.50_cuda11.6-archive/include/ -L /users/abaranwa/cudnn-linux-x86_64-8.4.1.50_cuda11.6-archive/lib64/ -lcudnn