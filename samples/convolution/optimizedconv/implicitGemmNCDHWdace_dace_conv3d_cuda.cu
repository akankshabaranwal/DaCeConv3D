#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct implicitGemmNCDHWdace_dace_conv3d_t {
    dace::cuda::Context *gpu_context;
};

// const float r_DHW=(1.0f/(32*32*32));
// const float r_HW=(1.0f/(32*32));
// const float r_W=(1.0f/32);

// const float r_DHW=(1.0f/(16*16*16));
// const float r_HW=(1.0f/(16*16));
// const float r_W=(1.0f/16);

const float r_DHW=(1.0f/(8*8*8));
const float r_HW=(1.0f/(8*8));
const float r_W=(1.0f/8);

// const float r_DHW=(1.0f/(4*4*4));
// const float r_HW=(1.0f/(4*4));
// const float r_W=(1.0f/4);

// const float r_DHW=(1.0f/(2*2*2));
// const float r_HW=(1.0f/(2*2));
// const float r_W=(1.0f/2);

const float r_kdim = (1.0f/3);
const float r_kdim2 = (1.0f/9);
const double r_kdim3 = (1.0f/27);
 
DACE_EXPORTED int __dace_init_cuda(implicitGemmNCDHWdace_dace_conv3d_t *__state, int d_batchsize, int d_inchannels, int d_kdim, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth);
DACE_EXPORTED void __dace_exit_cuda(implicitGemmNCDHWdace_dace_conv3d_t *__state);

DACE_DFI void implicitGemmNCDHWdace_dace_conv3d_118_4_125_12_128_16_129_20_130_24_0_0_27(const float* __restrict__ __tmp_153_66_r, const float* __restrict__ __tmp_154_67_r, float&  __tmp_153_28_w, float&  __tmp_154_28_w, int cta_k, int cta_m, int cta_n, int d_DHW, int d_HW, int d_batchsize, int d_inchannels, int d_kdim, int d_kdim2, int d_kdim3, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth, int gemm_m, int gemm_n, int warp_k, int warp_m, int warp_n) {
    float __tmp38[1]  DACE_ALIGN(64);
    int n;
    int nopq_residual;
    int o;
    int opq_residual;
    int p;
    int q;
    int c;
    int ctrs_residual;
    int t;
    int trs_residual;
    int r;
    int s;
    int __tmp19;
    int __tmp23;
    int __tmp29;
    int __tmp33;
    int d;
    int h;
    int w;
    int __sym___tmp15;
    int __sym___tmp25;
    int __sym___tmp28;
    int __sym___tmp32;
    int __sym___tmp34;

    {
        int __tmp17;

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            //__out = int((((cta_m + gemm_m) + warp_m) / d_DHW));
            __out = int((((cta_m + gemm_m) + warp_m) * r_DHW));
            ///////////////////

            n = __out;
        }
        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int((((cta_m + gemm_m) + warp_m) % d_DHW));
            ///////////////////

            nopq_residual = __out;
        }
        {
            int __in1 = nopq_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_HW));
            __out =int((__in1) * (r_HW));
            ///////////////////

            __tmp17 = __out;
        }
        {
            int __inp = __tmp17;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            o = __out;
        }

    }
    __tmp19 = (nopq_residual % d_HW);
    {
        int __tmp21;

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp19);
            ///////////////////

            opq_residual = __out;
        }
        {
            int __in1 = opq_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_outwidth));
            __out =int((__in1) *r_W );
            ///////////////////

            __tmp21 = __out;
        }
        {
            int __inp = __tmp21;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            p = __out;
        }

    }
    __tmp23 = (opq_residual % d_outwidth);
    {

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp23);
            ///////////////////

            q = __out;
        }
        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            //__out = int(((cta_k + warp_k) / d_kdim3));
            //__out = int(((cta_k + warp_k) * (1.0f/27)));
            __out = int(((cta_k + warp_k) * r_kdim3));
            // if((cta_k+warp_k) > 27)
            //     printf("AB::%f, ", __out);
            //__out = int((double(cta_k + warp_k)*r_kdim3));
            ///////////////////

            c = __out;
        }
        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(((cta_k + warp_k) % d_kdim3));
            ///////////////////

            ctrs_residual = __out;
        }

    }
    __tmp29 = (ctrs_residual % d_kdim2);
    {
        int __tmp27;
        int __tmp31;

        {
            int __in1 = ctrs_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_kdim2));
            __out =int((__in1) * (r_kdim2));
            ///////////////////

            __tmp27 = __out;
        }
        {
            int __inp = __tmp27;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            t = __out;
        }
        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp29);
            ///////////////////

            trs_residual = __out;
        }
        {
            int __in1 = trs_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_kdim));
            __out =int((__in1) * (r_kdim));
            ///////////////////

            __tmp31 = __out;
        }
        {
            int __inp = __tmp31;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            r = __out;
        }

    }
    __tmp33 = (trs_residual % d_kdim);
    d = (o + t);
    h = (p + r);
    {

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp33);
            ///////////////////

            s = __out;
        }

    }
    w = (q + s);
    __sym___tmp15 = n;
    __sym___tmp25 = c;
    {


        dace::CopyND<float, 1, false, 1>::template ConstDst<1>::Copy(
        __tmp_153_66_r + ((((((((__sym___tmp15 * d_inchannels) * ((d_kdim + d_outdepth) - 1)) * ((d_kdim + d_outheight) - 1)) * ((d_kdim + d_outwidth) - 1)) + (((__sym___tmp25 * ((d_kdim + d_outdepth) - 1)) * ((d_kdim + d_outheight) - 1)) * ((d_kdim + d_outwidth) - 1))) + ((d * ((d_kdim + d_outheight) - 1)) * ((d_kdim + d_outwidth) - 1))) + (h * ((d_kdim + d_outwidth) - 1))) + w), __tmp38, 1);

    }
    __sym___tmp25 = c;
    __sym___tmp28 = t;
    __sym___tmp32 = r;
    __sym___tmp34 = s;
    {

        {
            float __inp = __tmp38[0];
            float __out;

            ///////////////////
            // Tasklet code (assign_153_28)
            __out = __inp;
            ///////////////////

            __tmp_153_28_w = __out;
        }
        {
            float __inp = __tmp_154_67_r[(((((((__sym___tmp25 * d_kdim) * d_kdim) * d_kdim) + ((__sym___tmp28 * d_kdim) * d_kdim)) + (__sym___tmp32 * d_kdim)) + __sym___tmp34) + ((((d_inchannels * d_kdim) * d_kdim) * d_kdim) * ((cta_n + gemm_n) + warp_n)))];
            float __out;

            ///////////////////
            // Tasklet code (assign_154_28)
            __out = __inp;
            ///////////////////

            __tmp_154_28_w = __out;
        }

    }
    
}

DACE_DFI void implicitGemmNCDHWdace_dace_conv3d_118_4_125_12_156_16_0_0_22(const float* __restrict__ __tmp_157_69_r, float * __restrict__ __tmp_161_53_r_in_from_1_0_in_from_3_0, float * __restrict__ __tmp_162_54_r_in_from_1_0_in_from_3_0, float *  __tmp_165_77_r_in_from_2_0_in_from_3_0, float *  __tmp_165_32_w_out_of_2_1_out_of_3_1, int d_batchsize, int d_inchannels, int d_kdim, int d_outdepth, int d_outheight, int d_outwidth) {

    {

        {
            for (auto warp_k = 0; warp_k < 8; warp_k += 1) {
                float __tmp_161_32_w_out_of_1_1[4]  DACE_ALIGN(64);
                float __tmp_162_32_w_out_of_1_1[8]  DACE_ALIGN(64);
                {
                    for (auto gemm_n = 0; gemm_n < 8; gemm_n += 1) {
                        for (auto gemm_m = 0; gemm_m < 4; gemm_m += 1) {
                            {
                                float __inp = __tmp_161_53_r_in_from_1_0_in_from_3_0[(gemm_m + (128 * warp_k))];
                                float __out;

                                ///////////////////
                                // Tasklet code (assign_161_32)
                                __out = __inp;
                                ///////////////////

                                __tmp_161_32_w_out_of_1_1[gemm_m] = __out;
                            }
                            {
                                float __inp = __tmp_162_54_r_in_from_1_0_in_from_3_0[(gemm_n + (32 * warp_k))];
                                float __out;

                                ///////////////////
                                // Tasklet code (assign_162_32)
                                __out = __inp;
                                ///////////////////

                                __tmp_162_32_w_out_of_1_1[gemm_n] = __out;
                            }
                        }
                    }
                }
                {
                    for (auto gemm_n = 0; gemm_n < 8; gemm_n += 1) {
                        for (auto gemm_m = 0; gemm_m < 4; gemm_m += 1) {
                            float __tmp17;
                            float __tmp18;
                            {
                                float __in1 = __tmp_161_32_w_out_of_1_1[gemm_m];
                                float __in2 = __tmp_162_32_w_out_of_1_1[gemm_n];
                                float __out;

                                ///////////////////
                                // Tasklet code (_Mult_)
                                __out = (__in1 * __in2);
                                ///////////////////

                                __tmp17 = __out;
                            }
                            {
                                float __in2 = __tmp17;
                                float __in1 = __tmp_165_77_r_in_from_2_0_in_from_3_0[(gemm_m + (128 * gemm_n))];
                                float __out;

                                ///////////////////
                                // Tasklet code (_Add_)
                                __out = (__in1 + __in2);
                                ///////////////////

                                __tmp18 = __out;
                            }
                            {
                                float __inp = __tmp18;
                                float __out;

                                ///////////////////
                                // Tasklet code (assign_165_32)
                                __out = __inp;
                                ///////////////////

                                __tmp_165_32_w_out_of_2_1_out_of_3_1[(gemm_m + (128 * gemm_n))] = __out;
                            }
                        }
                    }
                }
            }
        }

    }
    
}

DACE_DFI void implicitGemmNCDHWdace_dace_conv3d_118_4_167_12_168_16_0_0_15(const float&  __tmp_179_64_r, float* __restrict__ __tmp_179_20_w, int cta_m, int cta_n, int d_DHW, int d_HW, int d_batchsize, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth, int gemm_m, int gemm_n, int warp_m, int warp_n) {
    int n;
    int nopq_residual;
    int o;
    int opq_residual;
    int p;
    int q;
    int __tmp17;
    int __tmp21;
    int __sym___tmp13;
    int __sym___tmp16;
    int __sym___tmp20;
    int __sym___tmp22;

    {
        int __tmp15;

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            //__out = int((((cta_m + gemm_m) + warp_m) / d_DHW));
            __out = int((((cta_m + gemm_m) + warp_m) * r_DHW));
            ///////////////////

            n = __out;
        }
        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int((((cta_m + gemm_m) + warp_m) % d_DHW));
            ///////////////////

            nopq_residual = __out;
        }
        {
            int __in1 = nopq_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_HW));
            __out =int((__in1) * (r_HW));
            ///////////////////

            __tmp15 = __out;
        }
        {
            int __inp = __tmp15;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            o = __out;
        }

    }
    __tmp17 = (nopq_residual % d_HW);
    {
        int __tmp19;

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp17);
            ///////////////////

            opq_residual = __out;
        }
        {
            int __in1 = opq_residual;
            int __out;

            ///////////////////
            // Tasklet code (_Div_)
            //__out =int((__in1) / (d_outwidth));
            __out =int((__in1) * (r_W));
            ///////////////////

            __tmp19 = __out;
        }
        {
            int __inp = __tmp19;
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__inp);
            ///////////////////

            p = __out;
        }

    }
    __tmp21 = (opq_residual % d_outwidth);
    __sym___tmp13 = n;
    __sym___tmp16 = o;
    {

        {
            int __out;

            ///////////////////
            // Tasklet code (_convert_to_int32_)
            __out = int(__tmp21);
            ///////////////////

            q = __out;
        }

    }
    __sym___tmp20 = p;
    __sym___tmp22 = q;
    {

        {
            float __inp = __tmp_179_64_r;
            float __out;

            ///////////////////
            // Tasklet code (assign_179_20)
            __out = __inp;
            ///////////////////

            __tmp_179_20_w[((((((((__sym___tmp13 * d_outchannels) * d_outdepth) * d_outheight) * d_outwidth) + ((__sym___tmp16 * d_outheight) * d_outwidth)) + (__sym___tmp20 * d_outwidth)) + __sym___tmp22) + (((d_outdepth * d_outheight) * d_outwidth) * ((cta_n + gemm_n) + warp_n)))] = __out;
        }

    }
    
}



int __dace_init_cuda(implicitGemmNCDHWdace_dace_conv3d_t *__state, int d_batchsize, int d_inchannels, int d_kdim, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    

    __state->gpu_context = new dace::cuda::Context(2, 2);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(implicitGemmNCDHWdace_dace_conv3d_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void implicitGemmNCDHWdace_dace_conv3d_118_0_0_0(const float * __restrict__ Input, float * __restrict__ Output, const float * __restrict__ kernel, int d_batchsize, int d_inchannels, int d_kdim, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth) {
    {
        {
            int cta_m = (128 * blockIdx.x);
            int cta_n = (32 * blockIdx.y);
            __shared__ float cta_reducedk[4096];
            {
                {
                    {
                        int warp_m = (4 * threadIdx.x);
                        int warp_n = (8 * threadIdx.y);
                        {
                            {
                                {
                                    for (auto gemm_n = 0; gemm_n < 8; gemm_n += 1) {
                                        for (auto gemm_m = 0; gemm_m < 4; gemm_m += 1) {
                                            {
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (assign_122_24)
                                                __out = 0;
                                                ///////////////////

                                                cta_reducedk[(((gemm_m + (128 * gemm_n)) + warp_m) + (128 * warp_n))] = __out;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
            {
                for (auto cta_k = 0; cta_k < (((d_inchannels * d_kdim) * d_kdim) * d_kdim); cta_k += 8) {
                    __shared__ float __tmp_153_28_w_out_of_1_1_out_of_1_1_out_of_1_1[1024];
                    __shared__ float __tmp_154_28_w_out_of_1_1_out_of_1_1_out_of_1_1[256];
                    {
                        {
                            {
                                int warp_m = (4 * threadIdx.x);
                                int warp_n = (8 * threadIdx.y);
                                {
                                    {
                                        {
                                            for (auto warp_k = 0; warp_k < 8; warp_k += 1) {
                                                {
                                                    for (auto gemm_n = 0; gemm_n < 8; gemm_n += 1) {
                                                        for (auto gemm_m = 0; gemm_m < 4; gemm_m += 1) {
                                                            implicitGemmNCDHWdace_dace_conv3d_118_4_125_12_128_16_129_20_130_24_0_0_27(&Input[0], &kernel[0], __tmp_153_28_w_out_of_1_1_out_of_1_1_out_of_1_1[((gemm_m + (128 * warp_k)) + warp_m)], __tmp_154_28_w_out_of_1_1_out_of_1_1_out_of_1_1[((gemm_n + (32 * warp_k)) + warp_n)], cta_k, cta_m, cta_n, ((d_outdepth * d_outheight) * d_outwidth), (d_outheight * d_outwidth), d_batchsize, d_inchannels, d_kdim, (d_kdim * d_kdim), ((d_kdim * d_kdim) * d_kdim), d_outchannels, d_outdepth, d_outheight, d_outwidth, gemm_m, gemm_n, warp_k, warp_m, warp_n);
                                                        }
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                    __syncthreads();
                    {
                        {
                            {
                                int warp_m = (4 * threadIdx.x);
                                int warp_n = (8 * threadIdx.y);
                                {
                                    {
                                        implicitGemmNCDHWdace_dace_conv3d_118_4_125_12_156_16_0_0_22(&Input[0], &__tmp_153_28_w_out_of_1_1_out_of_1_1_out_of_1_1[warp_m], &__tmp_154_28_w_out_of_1_1_out_of_1_1_out_of_1_1[warp_n], &cta_reducedk[(warp_m + (128 * warp_n))], &cta_reducedk[(warp_m + (128 * warp_n))], d_batchsize, d_inchannels, d_kdim, d_outdepth, d_outheight, d_outwidth);
                                    }
                                }
                            }
                        }
                    }
                    __syncthreads();
                }
            }
            {
                {
                    {
                        int warp_m = (4 * threadIdx.x);
                        int warp_n = (8 * threadIdx.y);
                        {
                            {
                                {
                                    for (auto gemm_n = 0; gemm_n < 8; gemm_n += 1) {
                                        for (auto gemm_m = 0; gemm_m < 4; gemm_m += 1) {
                                            implicitGemmNCDHWdace_dace_conv3d_118_4_167_12_168_16_0_0_15(cta_reducedk[(((gemm_m + (128 * gemm_n)) + warp_m) + (128 * warp_n))], &Output[0], cta_m, cta_n, ((d_outdepth * d_outheight) * d_outwidth), (d_outheight * d_outwidth), d_batchsize, d_outchannels, d_outdepth, d_outheight, d_outwidth, gemm_m, gemm_n, warp_m, warp_n);
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_implicitGemmNCDHWdace_dace_conv3d_118_0_0_0(implicitGemmNCDHWdace_dace_conv3d_t *__state, const float * __restrict__ Input, float * __restrict__ Output, const float * __restrict__ kernel, int d_batchsize, int d_inchannels, int d_kdim, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth);
void __dace_runkernel_implicitGemmNCDHWdace_dace_conv3d_118_0_0_0(implicitGemmNCDHWdace_dace_conv3d_t *__state, const float * __restrict__ Input, float * __restrict__ Output, const float * __restrict__ kernel, int d_batchsize, int d_inchannels, int d_kdim, int d_outchannels, int d_outdepth, int d_outheight, int d_outwidth)
{
    //r_DHW = (1/(d_outdepth*d_outheight*d_outwidth));
    //r_HW = (1/(d_outheight*d_outwidth));
    //r_W = (1/(d_outwidth));

    //r_kdim = (1/d_kdim);
    //r_kdim2 = (1/(d_kdim*d_kdim));
    //r_kdim3 = (1/(d_kdim*d_kdim*d_kdim));

    void  *implicitGemmNCDHWdace_dace_conv3d_118_0_0_0_args[] = { (void *)&Input, (void *)&Output, (void *)&kernel, (void *)&d_batchsize, (void *)&d_inchannels, (void *)&d_kdim, (void *)&d_outchannels, (void *)&d_outdepth, (void *)&d_outheight, (void *)&d_outwidth };
    hipLaunchKernel((void*)implicitGemmNCDHWdace_dace_conv3d_118_0_0_0, dim3(int_ceil((((d_batchsize * d_outdepth) * d_outheight) * d_outwidth), 128), int_ceil(d_outchannels, 32), 1), dim3(32, 4, 1), implicitGemmNCDHWdace_dace_conv3d_118_0_0_0_args, 0, __state->gpu_context->streams[0]);
}

