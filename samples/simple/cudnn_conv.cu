// References:
// http://courses.cms.caltech.edu/cs101gpu/2022_lectures/cs179_2022_lec17.pdf
// https://gist.github.com/odashi/1c20ba90388cf02330e1b95963d78039
// https://medium.com/@rohitdwivedula/minimal-cudnn-c-hello-world-example-47d3c6b60b73
// API Reference: https://docs.nvidia.com/deeplearning/cudnn/api/index.html#cudnnConvolutionForward
// https://gist.github.com/goldsborough/865e6717e64fbae75cdaf6c9914a130d

#include <iomanip>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <cassert>
#include <iostream>
#include <algorithm>

using namespace std;

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  cudnnStatus_t err = (f); \
  if (err != CUDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << cudnnGetErrorString(err)<<' '<<__LINE__<<std::endl; \
    std::exit(1); \
  } \
}

int main()
{
        int numGPUs;
        hipGetDeviceCount(&numGPUs);
        std::cout << "Found " << numGPUs << " GPUs." << std::endl;
        hipSetDevice(0); // use GPU0
        int device; 
        struct hipDeviceProp_t devProp;
        hipGetDevice(&device);
        hipGetDeviceProperties(&devProp, device);
        std::cout << "Compute capability:" << devProp.major << "." << devProp.minor << std::endl;

        cudnnHandle_t cudnn;
        CUDNN_CALL(cudnnCreate(&cudnn));
        std::cout << "Created cuDNN handle" << std::endl;

        // input
        const int in_n = 1, in_c = 1, in_d = 5, in_h = 5, in_w = 5;
        std::cout << "in_n: " << in_n << ", in_c: " << in_c << ", in_d: " << in_d << ", in_h: " << in_h << ", in_w: " << in_w << std::endl;
        cudnnTensorDescriptor_t in_desc;
        CUDNN_CALL(cudnnCreateTensorDescriptor(&in_desc));        
        vector<int> dims = {in_n, in_c, in_d, in_h, in_w};
        vector<int> strides = {in_c*in_d*in_h*in_w, in_d*in_h*in_w, in_h*in_w, in_w, 1};
        CUDNN_CALL(cudnnSetTensorNdDescriptor(in_desc, 
                                            CUDNN_DATA_FLOAT, 
                                            5, 
                                            dims.data(), 
                                            strides.data())
                                            );
        float *in_data;
        CUDA_CALL(hipMalloc( &in_data, in_n * in_c * in_d * in_h * in_w * sizeof(float)));

        // filter
        const int filt_k = 1, filt_c = 1, filt_d = 3, filt_h = 3, filt_w = 3;
        std::cout << "filt_k: " << filt_k << ", filt_c: " << filt_c << ", filt_d: " << filt_d << ", filt_h: " << filt_h << ", filt_w: " << filt_w << std::endl;
        cudnnFilterDescriptor_t filt_desc;
        CUDNN_CALL(cudnnCreateFilterDescriptor(&filt_desc));
        vector<int> filtdims = {filt_k, in_c, in_d, in_h, in_w};
        CUDNN_CALL(cudnnSetFilterNdDescriptor(filt_desc, 
                                            CUDNN_DATA_FLOAT, 
                                            CUDNN_TENSOR_NCHW, 
                                            5, 
                                            filtdims.data()));
        float *filt_data;
        CUDA_CALL(hipMalloc(&filt_data, filt_k * filt_c * filt_d * filt_h * filt_w * sizeof(float)));

        // convolution
        const int pad_d = 1, pad_h = 1, pad_w = 1, str_d=1, str_h = 1, str_w = 1, dil_d=1, dil_h = 1, dil_w = 1;
        std::cout << "pad_d: " << pad_d << ", pad_h: " << pad_h << ", pad_w: " << pad_w << ", str_d: " << str_d << ", str_h: " << str_h << ", str_w: " << str_w << ", dil_d: " << dil_d <<", dil_h: " << dil_h << ", dil_w: " << dil_w << std::endl;
        cudnnConvolutionDescriptor_t conv_desc;
        CUDNN_CALL(cudnnCreateConvolutionDescriptor(&conv_desc));
        vector<int> convpad = {pad_d, pad_h, pad_w};
        vector<int> filtstr = {str_d, str_h, str_w};
        vector<int> convdil = {dil_d, dil_h, dil_w};
        CUDNN_CALL(cudnnSetConvolutionNdDescriptor(conv_desc, 
                                                  3, 
                                                  convpad.data(), 
                                                  filtstr.data(), 
                                                  convdil.data(), 
                                                  CUDNN_CROSS_CORRELATION, 
                                                  CUDNN_DATA_FLOAT)
                                                  );

        // output
        int outdims[5];
        CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim( conv_desc, in_desc, filt_desc, 5, outdims));
        cudnnTensorDescriptor_t out_desc;        
        CUDNN_CALL(cudnnCreateTensorDescriptor(&out_desc));
        int out_n = outdims[0];
        int out_c = outdims[1];
        int out_d = outdims[2];
        int out_h = outdims[3];
        int out_w = outdims[4];
        std::cout << "out_n: " << out_n << ", out_c: " << out_c << ", out_d: "<< out_d<< ", out_h: " << out_h << ", out_w: " << out_w << std::endl;
        vector<int> outstrides = {out_c*out_d*out_h*out_w, out_d*out_h*out_w, out_h*out_w, out_w, 1};
        
        CUDNN_CALL(cudnnSetTensorNdDescriptor(out_desc, 
                                            CUDNN_DATA_FLOAT, 
                                            5, 
                                            outdims, 
                                            outstrides.data())
                                            );
        float *out_data;
        CUDA_CALL(hipMalloc(&out_data, out_n * out_c * out_d * out_h * out_w * sizeof(float)));


        void *search_ws;
        hipMalloc(&search_ws, 33554432);        
        cudnnConvolutionFwdAlgoPerf_t perfResults;
        int requestedAlgoCount = 1;
        int returnedAlgoCount = 1;
        CUDNN_CALL(cudnnFindConvolutionForwardAlgorithmEx(cudnn, in_desc, in_data, filt_desc, filt_data, conv_desc, 
                                                        out_desc, out_data, requestedAlgoCount, &returnedAlgoCount, &perfResults, search_ws, 33554432));
      // Till here the code works.
        assert(in_desc!=nullptr);
        assert(filt_desc!=nullptr);
        assert(out_desc!=nullptr);
        assert(conv_desc!=nullptr);

        hipFree(search_ws);
        cudnnConvolutionFwdAlgo_t selectedAlgo;
        selectedAlgo = perfResults.algo;
        
        std::cout<<selectedAlgo;
        size_t ws_size=33554432;
        CUDNN_CALL(cudnnGetConvolutionForwardWorkspaceSize(cudnn, in_desc, filt_desc, conv_desc, out_desc, selectedAlgo, &ws_size));
        std::cerr << "Workspace size: " << (ws_size ) << "bytes"<< std::endl;
        
        void* d_workspace{nullptr};
        hipMalloc(&d_workspace, ws_size);
        const float alpha = 1.0f, beta = 0.0f;
        CUDNN_CALL(cudnnConvolutionForward(cudnn, &alpha, in_desc, in_data, filt_desc, filt_data, conv_desc, selectedAlgo, d_workspace, ws_size, &beta, out_desc, out_data));
        hipFree(in_data);
        hipFree(out_data);
        hipFree(filt_data);
        hipFree(d_workspace);
        cudnnDestroyTensorDescriptor(in_desc);
        cudnnDestroyTensorDescriptor(out_desc);
        cudnnDestroyFilterDescriptor(filt_desc);
        cudnnDestroyConvolutionDescriptor(conv_desc);
}

// Command line to compile: nvcc cudnn_conv.cu -I /users/abaranwa/cudnn-linux-x86_64-8.4.1.50_cuda11.6-archive/include/ -L /users/abaranwa/cudnn-linux-x86_64-8.4.1.50_cuda11.6-archive/lib64/ -lcudnn