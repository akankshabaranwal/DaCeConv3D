#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct dace_simple_0_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(dace_simple_0_t *__state);
DACE_EXPORTED void __dace_exit_cuda(dace_simple_0_t *__state);



int __dace_init_cuda(dace_simple_0_t *__state) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    __state->gpu_context = new dace::cuda::Context(2, 1);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(dace_simple_0_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void dace_simple_78_d_0_0_21(const float * __restrict__ gpu_Input, float * __restrict__ gpu_Output, const float * __restrict__ gpu_kernel) {
    {
        {
            {
                int w = blockIdx.x;
                int h = blockIdx.y;
                int d = blockIdx.z;
                {
                    {
                        float r_tmp[1]  DACE_ALIGN(64);
                        int oc = threadIdx.x;
                        {
                            {
                                for (auto __i0 = 0; __i0 < 1; __i0 += 1) {
                                    {
                                        float __out;

                                        ///////////////////
                                        // Tasklet code (_numpy_full_)
                                        __out = 0.0;
                                        ///////////////////

                                        r_tmp[__i0] = __out;
                                    }
                                }
                            }
                            {
                                for (auto kd = 0; kd < 3; kd += 1) {
                                    for (auto kh = 0; kh < 3; kh += 1) {
                                        for (auto kw = 0; kw < 3; kw += 1) {
                                            for (auto ic = 0; ic < 4; ic += 1) {
                                                float __tmp4;
                                                float __tmp5;
                                                {
                                                    float __in2 = gpu_kernel[(((((8 * ic) + (288 * kd)) + (96 * kh)) + (32 * kw)) + oc)];
                                                    float __in1 = gpu_Input[(((((((4096 * d) + (128 * h)) + ic) + (4096 * kd)) + (128 * kh)) + (4 * kw)) + (4 * w))];
                                                    float __out;

                                                    ///////////////////
                                                    // Tasklet code (_Mult_)
                                                    __out = (__in1 * __in2);
                                                    ///////////////////

                                                    __tmp4 = __out;
                                                }
                                                {
                                                    float __in2 = __tmp4;
                                                    float __in1 = r_tmp[0];
                                                    float __out;

                                                    ///////////////////
                                                    // Tasklet code (_Add_)
                                                    __out = (__in1 + __in2);
                                                    ///////////////////

                                                    __tmp5 = __out;
                                                }
                                                {
                                                    float __inp = __tmp5;
                                                    float __out;

                                                    ///////////////////
                                                    // Tasklet code (assign_82_12)
                                                    __out = __inp;
                                                    ///////////////////

                                                    r_tmp[0] = __out;
                                                }
                                            }
                                        }
                                    }
                                }
                            }
                            {
                                float __inp = r_tmp[0];
                                float __out;

                                ///////////////////
                                // Tasklet code (assign_83_8)
                                __out = __inp;
                                ///////////////////

                                gpu_Output[((((8192 * d) + (256 * h)) + oc) + (8 * w))] = __out;
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_dace_simple_78_d_0_0_21(dace_simple_0_t *__state, const float * __restrict__ gpu_Input, float * __restrict__ gpu_Output, const float * __restrict__ gpu_kernel);
void __dace_runkernel_dace_simple_78_d_0_0_21(dace_simple_0_t *__state, const float * __restrict__ gpu_Input, float * __restrict__ gpu_Output, const float * __restrict__ gpu_kernel)
{

    void  *dace_simple_78_d_0_0_21_args[] = { (void *)&gpu_Input, (void *)&gpu_Output, (void *)&gpu_kernel };
    hipLaunchKernel((void*)dace_simple_78_d_0_0_21, dim3(int_ceil(30, 1), int_ceil(30, 1), int_ceil(30, 1)), dim3(8, 1, 1), dace_simple_78_d_0_0_21_args, 0, __state->gpu_context->streams[0]);
}

